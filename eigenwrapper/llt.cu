
#include <hip/hip_runtime.h>
/*#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <cuda_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_cuda.h>
#include <helper_functions.h>
#include "device_launch_parameters.h"
#define __blocksize 8
#define __chunk 128

#define __blocksize2 4
#define __chunk2 4

__global__ void __add(double* value, int* row, int* col, int N, int M, double* value2, int* index) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j * __chunk;
	int _e = _s + __chunk;
	if (_s >= _e)return;
	if (_e > N)_e = N;
	int* _row = row + _s;
	for (int i = _s; i < _e; i++)
	{
		int S = *_row;// row[i];
		_row++;
		int E =* _row;// row[i + 1];
		int __row = i;
		int* _col = col + S;
		double* _val = value + S;
		for (int k = S; k < E; k++)
		{
			int __col = *_col;// col[k];
			double __val = *_val;;// value[k];
			int __index = index[__row * M + __col];
			value2[__index] += __val;
			_val++;
			_col++;
		}		
	}
}
__global__ void __vecmul(double* value, int* row, int* col, int N, int M, double* value2) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j * __chunk;
	int _e = _s + __chunk;
	if (_s >= _e)return;
	if (_e > N)_e = N;
	int* _row = row + _s;
	double* _val2 = &value2[_s];
	for (int i = _s; i < _e; i++)
	{
		double val2 = sqrt(*_val2);
		int S = *_row;// row[i];
		_row++;
		int E = *_row;// row[i + 1];
		int __row = i;
		int* _col = col + S;
		double* _val = value + S;
		for (int k = S; k < E; k++)
		{
			int __col = *_col;// col[k];
			(*_val) *= val2;
			_val++;
			_col++;
		}
		_val2++;
	}
}
void kernel(double* value,int* row,int* col,int N,int M,double* value2,int* index,cudaStream_t stream) {
	
	dim3 threads(__blocksize);
	int ff = N / __blocksize / __chunk;
	ff++;
	dim3 grid(ff);
	cudaStreamSynchronize(stream);
	__add <<<grid, threads, 0, stream >> > (value, row, col, N, M, value2,index);

}

void kernel2(double* value, int* row, int* col, int N, int M, double* value2, cudaStream_t stream) {

	dim3 threads(__blocksize2);
	int ff = N / __blocksize2 / __chunk2;
	ff++;
	dim3 grid(ff);
	cudaStreamSynchronize(stream);
	__vecmul<< <grid, threads, 0, stream >> > (value, row, col, N, M, value2);

}
*/