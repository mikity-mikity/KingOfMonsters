#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""
#define __blocksize 8
#define __chunk 128

#define __blocksize2 4
#define __chunk2 4
/*__global__ void cholesky1(double* A, double* L, int n) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j* __chunk;
	int _e = _s + __chunk;
	if (_s >= n)return;
	if (_e > n)_e = n;

	for (int j = _s; j < _e; j++) {

		double s = 0;
		double* _ptr = &L[j * n + 0];
		for (int k = 0; k < j; k++) {
			s += *_ptr * *_ptr;
			_ptr++;
		}
		L[j * n + j] = sqrt(A[j * n + j] - s);
	}
}
__global__ void cholesky1(double* A, double* L, int j, int n) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j * __chunk;
	int _e = _s + __chunk;
	if (_s >= _e)return;
	if (_e > n)_e = n;
	



	if(_s==0)
		L[j * n + j] += sqrt(A[j * n + j]);

	double s = 0;
	int __s2 = _s;
	int __e2 = _e;
	if (__e2 >= j)__e2 = j;
	if (__s2 < __e2)
	{
		double* _ptr = &L[j * n + __s2];
		for (int k = __s2; k < __e2; k++) {
			s += *_ptr * *_ptr;
			_ptr++;
		}
		L[j * n + j] -= s;
	}
	//L[j * n + j] = sqrt(A[j * n + j] - s);
	
	return;
	__s2 = _s;
	__e2 = _e;
	if (__s2 < j + 1)__s2 = j + 1;
	if (__e2 >= n)__e2 = n;
	if (__s2 < __e2)
	{
		for (int i = __s2; i < __e2; i++) {
			double s = 0;
			double* ptr = &L[i * n + 0];
			double* ptr2 = &L[j * n + 0];
			for (int k = 0; k < j; k++) {
				//s += L[i * n + k] * L[j * n + k];
				s += *ptr * *ptr2;
				ptr++;
				ptr2++;
			}
			if(_s==0)
				L[i * n + j] += (1.0 / L[j * n + j] * (A[i * n + j]));
			L[i * n + j] -= 1.0 / L[j * n + j] * s;
		}

	}
}*/
__global__ void __add(double* value, int* row, int* col, int N, int M, double* value2, int* index) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j * __chunk;
	int _e = _s + __chunk;
	if (_s >= _e)return;
	if (_e > N)_e = N;
	int* _row = row + _s;
	for (int i = _s; i < _e; i++)
	{
		int S = *_row;// row[i];
		_row++;
		int E =* _row;// row[i + 1];
		int __row = i;
		int* _col = col + S;
		double* _val = value + S;
		for (int k = S; k < E; k++)
		{
			int __col = *_col;// col[k];
			double __val = *_val;;// value[k];
			int __index = index[__row * M + __col];
			value2[__index] += __val;
			_val++;
			_col++;
		}		
	}
}
__global__ void __vecmul(double* value, int* row, int* col, int N, int M, double* value2) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j * __chunk;
	int _e = _s + __chunk;
	if (_s >= _e)return;
	if (_e > N)_e = N;
	int* _row = row + _s;
	double* _val2 = &value2[_s];
	for (int i = _s; i < _e; i++)
	{
		double val2 = sqrt(*_val2);
		int S = *_row;// row[i];
		_row++;
		int E = *_row;// row[i + 1];
		int __row = i;
		int* _col = col + S;
		double* _val = value + S;
		for (int k = S; k < E; k++)
		{
			int __col = *_col;// col[k];
			(*_val) *= val2;
			_val++;
			_col++;
		}
		_val2++;
	}
}
void kernel(double* value,int* row,int* col,int N,int M,double* value2,int* index,hipStream_t stream) {
	
	dim3 threads(__blocksize);
	int ff = N / __blocksize / __chunk;
	ff++;
	dim3 grid(ff);
	hipStreamSynchronize(stream);
	__add <<<grid, threads, 0, stream >> > (value, row, col, N, M, value2,index);

}

void kernel2(double* value, int* row, int* col, int N, int M, double* value2, hipStream_t stream) {

	dim3 threads(__blocksize2);
	int ff = N / __blocksize2 / __chunk2;
	ff++;
	dim3 grid(ff);
	hipStreamSynchronize(stream);
	__vecmul<< <grid, threads, 0, stream >> > (value, row, col, N, M, value2);

}