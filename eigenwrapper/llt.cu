#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""
#define __blocksize 8
#define __chunk 8
__global__ void cholesky4(double* A, double* L, int n) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j* __chunk;
	int _e = _s + __chunk;
	if (_s >= n)return;
	if (_e > n)_e = n;



	int _b2 = blockIdx.y;
	int _t2 = threadIdx.y;
	int _j2 = _b2 * __blocksize + _t2;
	int _s2 = _j2 * __chunk;
	int _e2 = _s2 + __chunk;
	if (_s2 >= n)return;
	if (_s2 >= _e2)return;
	for (int j = _s; j < _e; j++) {

		double s = 0;
		double* _ptr = &L[j * n + 0];
		for (int k = 0; k < j; k++) {
			s += *_ptr * *_ptr;
			_ptr++;
		}
		L[j * n + j] = sqrt(A[j * n + j] - s);
		int __s2 = 0;
		int __e2 = 0;
		if (_s2 >= j + 1)__s2 = _s2; else __s2 = j + 1;
		if (_e2 <= n)__e2 = _e2; else __e2 = n;
		if (__s2 >= __e2)return;
			for (int i = __s2; i < __e2; i++) {
				double s = 0;
				double* ptr = &L[i * n + 0];
				double* ptr2 = &L[j * n + 0];
				for (int k = 0; k < j; k++) {
					//s += L[i * n + k] * L[j * n + k];
					s += *ptr * *ptr2;
					ptr++;
					ptr2++;
				}
				L[i * n + j] = (1.0 / L[j * n + j] * (A[i * n + j] - s));
			}

	}
}


void kernel(double* A,double *work, int N,hipStream_t stream) {
	
	dim3 threads(__blocksize, __blocksize);
	int ff = N / __blocksize / __chunk;
	if (ff == 0)ff = 1;
	dim3 grid(ff,ff);

	cholesky4<<<grid, threads,0,stream>>>(A, work, N);

}