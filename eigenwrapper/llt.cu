#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""
#define __blocksize 8
#define __chunk 4
__global__ void cholesky1(double* A, double* L, int n) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j* __chunk;
	int _e = _s + __chunk;
	if (_s >= n)return;
	if (_e > n)_e = n;

	for (int j = _s; j < _e; j++) {

		double s = 0;
		double* _ptr = &L[j * n + 0];
		for (int k = 0; k < j; k++) {
			s += *_ptr * *_ptr;
			_ptr++;
		}
		L[j * n + j] = sqrt(A[j * n + j] - s);
	}
}
__global__ void cholesky1(double* A, double* L, int j, int n) {
	int _b = blockIdx.x;
	int _t = threadIdx.x;
	int _j = _b * __blocksize + _t;
	int _s = _j * __chunk;
	int _e = _s + __chunk;
	if (_s >= _e)return;
	if (_e > n)_e = n;
	


	/*int _b2 = blockIdx.y;
	int _t2 = threadIdx.y;
	int _j2 = _b2 * __blocksize + _t2;
	int _s2 = _j2 * __chunk;
	int _e2 = _s2 + __chunk;
	if (_s2 >= n)return;
	if (_s2 >= _e2)return;*/
	if(_s==0)
		L[j * n + j] += sqrt(A[j * n + j]);

	double s = 0;
	int __s2 = _s;
	int __e2 = _e;
	if (__e2 >= j)__e2 = j;
	if (__s2 < __e2)
	{
		double* _ptr = &L[j * n + __s2];
		for (int k = __s2; k < __e2; k++) {
			s += *_ptr * *_ptr;
			_ptr++;
		}
		L[j * n + j] -= s;
	}
	//L[j * n + j] = sqrt(A[j * n + j] - s);
	/*int __s2 = _s2;
	int __e2 = _e2;
	//__s2 = j + 1 + _s2 * (n - j - 1) / n;
	//__e2 = j + 1 + _e2 * (n - j - 1) / n;
	if (_s2 >= j + 1)__s2 = _s2; else __s2 = j + 1;
	if (_e2 <= n)__e2 = _e2; else __e2 = n;
	if (__s2 >= __e2)return;
	*/
	return;
	__s2 = _s;
	__e2 = _e;
	if (__s2 < j + 1)__s2 = j + 1;
	if (__e2 >= n)__e2 = n;
	if (__s2 < __e2)
	{
		for (int i = __s2; i < __e2; i++) {
			double s = 0;
			double* ptr = &L[i * n + 0];
			double* ptr2 = &L[j * n + 0];
			for (int k = 0; k < j; k++) {
				//s += L[i * n + k] * L[j * n + k];
				s += *ptr * *ptr2;
				ptr++;
				ptr2++;
			}
			if(_s==0)
				L[i * n + j] += (1.0 / L[j * n + j] * (A[i * n + j]));
			L[i * n + j] -= 1.0 / L[j * n + j] * s;
		}

	}
}

void kernel(double* A,double *work, int N,hipStream_t stream) {
	
	dim3 threads(__blocksize);
	int ff = N / __blocksize / __chunk;
	ff++;
	dim3 grid(ff);
	hipMemsetAsync(work, 0, sizeof(double) * N * N, stream);
	hipStreamSynchronize(stream);
	for (int j = 0; j < N; j++) {
		cholesky1<<<grid, threads,0,stream>>>(A, work, j,N);
	}


	//cholesky4<<<grid, threads,0,stream>>>(A, work, N);

}